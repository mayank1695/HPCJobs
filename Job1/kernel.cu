
#include <hip/hip_runtime.h>
//No headers
//ncvv automatically imports the required
__global__ void process_kernel1(float *input1,float *input2,float *output,int datasize){
	int blockNum = blockIdx.z * (gridDim.x *gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x*blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	int n = datasize/sizeof(input1[0]);
	if(id<n)output[id] = sin(input1[id]) + cos(input2[id]);
}

__global__ void process_kernel2(float *input,float *output,int datasize){
	int blockNum = blockIdx.z * (gridDim.x *gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x*blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	int n = datasize/sizeof(input[0]);
	if(id<n)output[id] = log(input[id]);
}

__global__ void process_kernel3(float *input,float *output,int datasize){
	int blockNum = blockIdx.z * (gridDim.x *gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x*blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	int n = datasize/sizeof(input[0]);
	if(id<n)output[id] = sqrt(input[id]);
}

