#include "hip/hip_runtime.h"
#include<job1.h>

void prepHostMem(void **,int,size_t);
void prepDeviceMem(void **,void **,int,int);
void freeDeviceMem(float **,int);
void kernel1Run(float **,float **);
void kernel2Run(float **,float **);
void kernel3Run(float **,float **);

/**
 * Host main Routine
 */
int main(void){
	int numElements = 50000;
	size_t size = numElements * sizeof(float);
	hipError_t err;
	float **list_host,**list_device;
	
	/*************************************************************************************************
	* process_kernel1 segment 
	**************************************************************************************************/
	printf("Preparing for process_kernel1.\n");
	
	/*****************************************************************************/
	//Host memory preparation
	list_host = (float**)malloc(3*sizeof(float*));
	prepHostMem((void**)list_host,3,size);
	for(int a=0;a<3;a++)list_host[a]=(float*)list_host[a];
	// Initialize the host input vectors
    	for (int i = 0; i < numElements; i++){
        	list_host[0][i] = rand()/(float)RAND_MAX;
	        list_host[1][i] = rand()/(float)RAND_MAX;
    	}
	/*****************************************************************************/

	/*****************************************************************************/	
	//Device arrays
	list_device=(float**)malloc(3*sizeof(float*));
	prepDeviceMem((void**)list_host,(void**)list_device,3,2);
	/*****************************************************************************/
    	
	/*****************************************************************************/    	
	//process_kernel1	
	kernel1Run(list_host,list_device);
	//Freeing host and device list
	free(list_host);
	free(list_device);
	printf("Job process_kernel1 finished.\n");
	/*****************************************************************************/    	
	
	
	/*************************************************************************************************
	* process_kernel2 segment 
	**************************************************************************************************/
	printf("Preparing for process_kernel2.\n");
	
	/*****************************************************************************/
	//Host memory preparation
	list_host = (float**)malloc(2*sizeof(float*));
	prepHostMem((void**)list_host,2,size);
	for(int a=0;a<2;a++)list_host[a]=(float*)list_host[a];
	// Initialize the host input vectors
    	for (int i = 0; i < numElements; i++){
        	list_host[0][i] = rand()/(float)RAND_MAX;
    	}
	/*****************************************************************************/

	/*****************************************************************************/	
	//Device arrays
	list_device=(float**)malloc(2*sizeof(float*));
	prepDeviceMem((void**)list_host,(void**)list_device,2,1);
	/*****************************************************************************/
    	
	/*****************************************************************************/    	
	//process_kernel1	
	kernel2Run(list_host,list_device);
	//Freeing host and device list
	free(list_host);
	free(list_device);
	printf("Job process_kernel2 finished.\n");
	/*****************************************************************************/

	/*************************************************************************************************
	* process_kernel3 segment 
	**************************************************************************************************/
	printf("Preparing for process_kernel3.\n");
	
	/*****************************************************************************/
	//Host memory preparation
	list_host = (float**)malloc(2*sizeof(float*));
	prepHostMem((void**)list_host,2,size);
	for(int a=0;a<2;a++)list_host[a]=(float*)list_host[a];
	// Initialize the host input vectors
    	for (int i = 0; i < numElements; i++){
        	list_host[0][i] = rand()/(float)RAND_MAX;
    	}
	/*****************************************************************************/

	/*****************************************************************************/	
	//Device arrays
	list_device=(float**)malloc(2*sizeof(float*));
	prepDeviceMem((void**)list_host,(void**)list_device,2,1);
	/*****************************************************************************/
    	
	/*****************************************************************************/    	
	//process_kernel1	
	kernel3Run(list_host,list_device);
	//Freeing host and device list
	free(list_host);
	free(list_device);
	printf("Job process_kernel3 finished.\n");
	/*****************************************************************************/    	


	/*****************************************************************************/
    	// Reset the device and exit
    	// hipDeviceReset causes the driver to clean up all state. While
    	// not mandatory in normal operation, it is good practice.  It is also
    	// needed to ensure correct operation when the application is being
    	// profiled. Calling hipDeviceReset causes all profile data to be
    	// flushed before the application exits
    	err = hipDeviceReset();

    	if (err != hipSuccess){
        	fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
	/*****************************************************************************/
    	printf("All jobs finished.\n");
	fflush(stdout);
    	return 0;
}
/**
 *This function mallocs a list of host arrays each of size "size".
 *The list is supposed to be of length "l".
 */
void prepHostMem(void **list_host,int l,size_t size){
	printf("Allocating host memory...");
	for(int a=0;a<l;a++){
		list_host[a] = malloc(size);
		if (list_host[a]==NULL){
			fprintf(stderr,"Failed to allocate host vectors!\n");
			printf("Fail.\n");
			exit(EXIT_FAILURE);
		}
	}
	printf("OK.\n");
}
/**
 *This function mallocs a list of device arrays each corresponding to a host array.
 *The function not only cudaMallocs, but also copies memory from host array to device array.
 *Thus it takes list of host and device arrays. It cudaMallocs "l1" number of device arrays
 *and copies "l2" number of host arrays to device arrays.
 */
void prepDeviceMem(void **list_host,void **list_device,int l1,int l2){
	printf("Preparing device memory:\n");
	hipError_t err = hipSuccess;
	printf("Allcating device memory...");
	//Allocating device memory
	for(int a=0;a<l1;a++){
		err = hipMalloc((void **)&list_device[a], sizeof(list_host[a]));
    		if (err != hipSuccess){
        		fprintf(stderr, "Failed to allocate device vector %d (error code %s)!\n", a+1,hipGetErrorString(err));
			printf("Fail.\n");
        		exit(EXIT_FAILURE);
    		}	
	}
	printf("OK.\n");
	//Copying from host to device
	printf("Copying to device memory...");
	//Allocating device memory
	for(int a=0;a<l2;a++){
		err = hipMemcpy(list_device[a], list_host[a], sizeof(list_host[a]), hipMemcpyHostToDevice);
    		if (err != hipSuccess){
        		fprintf(stderr, "Failed to copy vector %d from host to device (error code %s)!\n", a+1, hipGetErrorString(err));
			printf("Fail.\n");
        		exit(EXIT_FAILURE);
    		}
	}
	printf("OK.\n");
}
/**
 *This function frees a list of device arrays.
 *The list is supposed to be of length "l".
 */
void freeDeviceMem(float **list_device,int l){
	printf("Freeing device memory...");
	hipError_t err;
	for(int a=0;a<l;a++){
		err = hipFree(list_device[a]);
    		if (err != hipSuccess){
        		fprintf(stderr, "Failed to free device vector %d (error code %s)!\n", a+1, hipGetErrorString(err));
			printf("Fail.\n");
        		exit(EXIT_FAILURE);
    		}
	}
	printf("OK.\n");
}
/**
 *This function encapsulates the run of process_kernel1.
 *It takes the list of host and device arrays required to run.
 *This functions completes the GPU run and also does the testing part.
 */
void kernel1Run(float **list_host,float **list_device){
	printf("CUDA kernel launch with (4,2,2) blocks of (32,32,1) threads.\n");
	hipError_t err;
	int l=3;
	int size = sizeof(list_device[0]);
	int numElements = size/sizeof(float);
	dim3 X(4,2,2);
	dim3 Y(32,32,1);
	process_kernel1<<<X, Y>>>(list_device[0], list_device[1], list_device[2], size);
    	err = hipGetLastError();
    	if (err != hipSuccess){
        	fprintf(stderr, "Failed to launch process1 kernel (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
	printf("Kernel successfully returned from device.\n");

    	// Copy the device result vector in device memory to the host result vector
    	// in host memory.
    	printf("Copying output data from the CUDA device to the host memory...");
    	err = hipMemcpy((void*)list_host[2], (void*)list_device[2], size, hipMemcpyDeviceToHost);
    	if (err != hipSuccess){
        	fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
		printf("Fail.\n");
        	exit(EXIT_FAILURE);
    	}
	printf("OK.\n");
    	// Verify that the result vector is correct
	printf("Verifying results...");
    	for (int i = 0; i < numElements; ++i){
        	if (fabs(sin(list_host[0][i]) + cos(list_host[1][i]) - list_host[2][i]) > 1e-5){
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			printf("Fail.\n");
        	    	exit(EXIT_FAILURE);
        	}
    	}
    	printf("OK [Test Passed].\n");

	//Free device global memory
	freeDeviceMem(list_device,l);

    	// Free host memory
	printf("Freeing host memory...");
    	for(int a=0;a<l;a++)free(list_host[a]);
	printf("OK.\n");
}
/**
 *This function encapsulates the run of process_kernel2.
 *It takes the list of host and device arrays required to run.
 *This functions completes the GPU run and also does the testing part.
 */
void kernel2Run(float **list_host,float **list_device){
	int size = sizeof(list_device[0]);
	int numElements = size/sizeof(float);
	int blockz = 16;
	int gridy = (int)ceil((float)numElements/(8*8*blockz));
	printf("CUDA kernel launch with (2,%d,1) blocks of (8,8,%d) threads.\n",gridy,blockz);
	hipError_t err;
	int l=2;
	dim3 X(2,gridy,1);
	dim3 Y(8,8,blockz);
	process_kernel2<<<X, Y>>>(list_device[0], list_device[1], size);
    	err = hipGetLastError();
    	if (err != hipSuccess){
        	fprintf(stderr, "Failed to launch process1 kernel (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
	printf("Kernel successfully returned from device.\n");

    	// Copy the device result vector in device memory to the host result vector
    	// in host memory.
    	printf("Copying output data from the CUDA device to the host memory...");
    	err = hipMemcpy((void*)list_host[1], (void*)list_device[1], size, hipMemcpyDeviceToHost);
    	if (err != hipSuccess){
        	fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
		printf("Fail.\n");
        	exit(EXIT_FAILURE);
    	}
	printf("OK.\n");
    	// Verify that the result vector is correct
	printf("Verifying results...");
    	for (int i = 0; i < numElements; ++i){
        	if (fabs(log(list_host[0][i]) - list_host[1][i]) > 1e-5){
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			printf("Fail.\n");
        	    	exit(EXIT_FAILURE);
        	}
    	}
    	printf("OK [Test Passed].\n");

	//Free device global memory
	freeDeviceMem(list_device,l);

    	// Free host memory
	printf("Freeing host memory...");
    	for(int a=0;a<l;a++)free(list_host[a]);
	printf("OK.\n");
}
/**
 *This function encapsulates the run of process_kernel3.
 *It takes the list of host and device arrays required to run.
 *This functions completes the GPU run and also does the testing part.
 */
void kernel3Run(float **list_host,float **list_device){
	int size = sizeof(list_device[0]);
	int numElements = size/sizeof(float);
	int blocky = 4;
	int gridx = (int)ceil((float)numElements/(128*blocky));
	printf("CUDA kernel launch with (%d,1,1) blocks of (128,%d,1) threads.\n",gridx,blocky);
	hipError_t err;
	int l=2;
	dim3 X(gridx,1,1);
	dim3 Y(128,blocky,1);
	process_kernel3<<<X, Y>>>(list_device[0], list_device[1], size);
    	err = hipGetLastError();
    	if (err != hipSuccess){
        	fprintf(stderr, "Failed to launch process1 kernel (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
	printf("Kernel successfully returned from device.\n");

    	// Copy the device result vector in device memory to the host result vector
    	// in host memory.
    	printf("Copying output data from the CUDA device to the host memory...");
    	err = hipMemcpy((void*)list_host[1], (void*)list_device[1], size, hipMemcpyDeviceToHost);
    	if (err != hipSuccess){
        	fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
		printf("Fail.\n");
        	exit(EXIT_FAILURE);
    	}
	printf("OK.\n");
    	// Verify that the result vector is correct
	printf("Verifying results...");
    	for (int i = 0; i < numElements; ++i){
        	if (fabs(sqrt(list_host[0][i]) - list_host[1][i]) > 1e-5){
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			printf("Fail.\n");
        	    	exit(EXIT_FAILURE);
        	}
    	}
    	printf("OK [Test Passed].\n");

	//Free device global memory
	freeDeviceMem(list_device,l);

    	// Free host memory
	printf("Freeing host memory...");
    	for(int a=0;a<l;a++)free(list_host[a]);
	printf("OK.\n");
}
